#include "hip/hip_runtime.h"
#include "CppCommon/WaveFrontDataStructs.h"
#include "CppCommon/RenderingUtility.h"

#include "../../vendor/Include/Cuda/cuda/helpers.h"
#include "../../vendor/Include/sutil/vec_math.h"
#include "../../vendor/Include/Optix/optix.h"
#include "../../vendor/Include/Optix/optix_device.h"

extern "C"
{

    __constant__ WaveFront::OptixLaunchParameters launchParams;

}



template<typename T>
__device__ __forceinline__ static T* UnpackPointer(unsigned int a_Upper, unsigned int a_Lower)
{

    const unsigned long long ptr = static_cast<unsigned long long>(a_Upper) << 32 | a_Lower;

    return reinterpret_cast<T*>(ptr);

}

template<typename T>
__device__ __forceinline__ static void PackPointer(T const* const a_Ptr, unsigned int& a_Upper, unsigned int& a_Lower)
{

    const unsigned long long ptr = reinterpret_cast<unsigned long long>(a_Ptr);

    a_Upper = ptr >> 32;
    a_Lower = ptr & 0x00000000ffffffff;

}



extern "C"
__device__ __forceinline__ void IntersectionRaysRayGen()
{


 
    //1. Get ray definition from buffer
    const unsigned idx = optixGetLaunchIndex().x;
    const WaveFront::IntersectionRayData& rayData = *launchParams.m_IntersectionRayBatch->GetData(idx);

    //2. Trace ray: optixTrace()
    WaveFront::IntersectionData intersection{};
    intersection.m_RayArrayIndex = idx;
    intersection.m_PixelIndex = rayData.m_PixelIndex;

    unsigned int intersectionPtr_Up = 0;
    unsigned int intersectionPtr_Low = 0;

    PackPointer(&intersection, intersectionPtr_Up, intersectionPtr_Low);

    const OptixTraversableHandle scene = launchParams.m_TraversableHandle;

    optixTrace(
        scene,
        rayData.m_Origin,
        rayData.m_Direction,
        launchParams.m_MinMaxDistance.x,
        launchParams.m_MinMaxDistance.y,
        0.f, //Ray Time, can be 0 in our case.
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_NONE,
        0, //SBT offset for selecting the SBT records to use
        0, //SBT stride for selecting the SBT records to use, multiplied with SBT-GAS index
        0, //Miss SBT index, always use first miss shader.
        intersectionPtr_Up,
        intersectionPtr_Low);

    //3. Store IntersectionData in buffer
    launchParams.m_IntersectionBuffer->Add(&intersection);

    return;

}

extern "C"
__device__ __forceinline__ void ShadowRaysRayGen()
{

    unsigned int idx = optixGetLaunchIndex().x;
    const WaveFront::ShadowRayData& rayData = *launchParams.m_ShadowRayBatch->GetData(idx);

    //2. Trace ray: optixTrace()
    const OptixTraversableHandle scene = launchParams.m_TraversableHandle;

    unsigned int isIntersection = 0;

    optixTrace(
        scene,
        rayData.m_Origin,
        rayData.m_Direction,
        launchParams.m_MinMaxDistance.x,
        rayData.m_MaxDistance,
        0.f,
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_NONE,
        0,
        1,
        0,
        isIntersection);

    //3. If no hit, accumulate result in buffer

    if(isIntersection == 0)
    {

        using namespace WaveFront;

        unsigned int resultIndex =
            static_cast<unsigned int>(LightChannel::NUM_CHANNELS) * rayData.m_PixelIndex +
            static_cast<unsigned int>(rayData.m_OutputChannel);

        launchParams.m_ResultBuffer[resultIndex] += rayData.m_PotentialRadiance;
    }

    return;

}

extern "C"
__device__ __forceinline__ void ReSTIRRayGen()
{
    //Launch as a 1D Array so that idx.x corresponds to the literal ray index.
    unsigned int idx = optixGetLaunchIndex().x;

    //Retrieve the data.
    const RestirShadowRay& rayData = *launchParams.m_ReSTIRShadowRayBatch->GetData(idx);
    const OptixTraversableHandle scene = launchParams.m_TraversableHandle;
    auto reservoirIndex = rayData.index;

    unsigned int intersected = 0;

    optixTrace(
        scene,
        rayData.origin,
        rayData.direction,
        launchParams.m_MinMaxDistance.x, //Prevent self shadowing so offset a little bit.
        rayData.distance,   //Max distance already has a small offset to prevent self-shadowing.
        0.f,
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_NONE,
        0,
        0,
        0,
        intersected  //Pass the reservoir index so that it can be set to 0 when a hit is found.
    );

    if(intersected != 0)
    {
        launchParams.m_Reservoirs[reservoirIndex].weight = 0.f;
    }
}



extern "C"
__global__ void __raygen__WaveFrontRG()
{

	switch (launchParams.m_TraceType)
	{
    case WaveFront::RayType::INTERSECTION_RAY:
        //Primary rays
        IntersectionRaysRayGen();
        break;
    case WaveFront::RayType::SHADOW_RAY:
        //Shadow rays
        ShadowRaysRayGen();
        break;
    case WaveFront::RayType::RESTIR_RAY:
        {
            //ReSTIR Rays.
            ReSTIRRayGen();
        }

        break;
	}

    return;

}


extern "C"
__global__ void __miss__WaveFrontMS()
{

    /*switch (launchParams.m_TraceType)
    {
    case WaveFront::RayType::INTERSECTION_RAY:
        return;
        break;
    case WaveFront::RayType::SHADOW_RAY:
        return;
        break;
    case WaveFront::RayType::RESTIR_RAY:
        return;
        break;
    }*/

    return;

}


extern "C"
__global__ void __anyhit__WaveFrontAH()
{

    switch (launchParams.m_TraceType)
    {
    case WaveFront::RayType::INTERSECTION_RAY:
        break;
    case WaveFront::RayType::SHADOW_RAY:
        {
            optixSetPayload_0(1);
            optixTerminateRay();
        }
        break;
    case WaveFront::RayType::RESTIR_RAY:
        {
            //Any hit is enough.
            optixSetPayload_0(1);
            optixTerminateRay();
        }
        break;
    }

    return;

}


extern "C"
__global__ void __closesthit__WaveFrontCH()
{

    switch (launchParams.m_TraceType)
    {
    case WaveFront::RayType::INTERSECTION_RAY:
        {
            //If closest hit found, return IntersectionData.
            const unsigned int intersectionPtr_Up = optixGetPayload_0();
            const unsigned int intersectionPtr_Low = optixGetPayload_1();
            WaveFront::IntersectionData* intersection = UnpackPointer<WaveFront::IntersectionData>(intersectionPtr_Up, intersectionPtr_Low);

            ////TODO: Try to fit this into 4 floats and one write.
            intersection->m_IntersectionT = optixGetRayTmax();
            intersection->m_Barycentrics = optixGetTriangleBarycentrics();
            intersection->m_PrimitiveIndex = optixGetPrimitiveIndex();
            intersection->m_InstanceId = optixGetInstanceId();

        }    
        break;
    case WaveFront::RayType::SHADOW_RAY:
        break;
    case WaveFront::RayType::RESTIR_RAY:
        //Get the reservoir and set its weight to 0 so that it is no longer considered a valid candidate.
        //reSTIRParams.reservoirs[optixGetAttribute_0()].weight = 0.f;
        //optixTerminateRay();
        break;
    }

    return;

}