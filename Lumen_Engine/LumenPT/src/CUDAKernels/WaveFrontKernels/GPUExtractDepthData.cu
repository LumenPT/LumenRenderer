#include "hip/hip_runtime.h"
#include "GPUShadingKernels.cuh"
#include "../../Shaders/CppCommon/Half4.h"

#include <>

CPU_ON_GPU void ExtractDepthDataGpu(
    const SurfaceData* a_SurfaceData,
    hipSurfaceObject_t a_DepthOutPut,
    uint2 a_Resolution,
    float2 a_MinMaxDistance)
{

    const unsigned int pixelX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int pixelY = blockIdx.y * blockDim.y + threadIdx.y;

    const unsigned int pixelDataIndex = PIXEL_DATA_INDEX(pixelX, pixelY, a_Resolution.x);

    //Check if a_OutPut->m_IntersectionT > a_DepthOutPut->depthValueAtPixelIndex to avoid writing to it if the T < value already there
    //if T > valueAtPixel ? overwrite : keep valueAtPixel

    if (pixelX < a_Resolution.x && pixelY < a_Resolution.y)
    {
        float t = a_SurfaceData[pixelDataIndex].m_IntersectionT;

        //float1 t = make_float1(a_SurfaceData[pixelDataIndex].m_IntersectionT);

        if (a_SurfaceData[pixelDataIndex].m_IntersectionT < 0.f)  //below 0 == no intersection
        {
            float1 nullResult = make_float1(0.f);

            surf2Dwrite<float1>(
                nullResult,
                a_DepthOutPut,  //intput
                pixelX * sizeof(float1),
                pixelY,
                hipBoundaryModeTrap);

            //half4Ushort4 nullResult = { make_float4(0.f, 0.f, 0.f, 0.f) };
            //surf2Dwrite<ushort4>(
            //    nullResult.m_Ushort4,
            //    a_DepthOutPut,  //intput
            //    pixelX * sizeof(ushort4),
            //    pixelY,
            //    hipBoundaryModeTrap);
            return;
        }

        t = (t - fminf(a_MinMaxDistance.x, t)) / (fmaxf(a_MinMaxDistance.y, t) - fminf(a_MinMaxDistance.x, t));

        //float tt = (t - fminf(0.f, t)) / (fmaxf(1.f, t) - fminf(0.f, t));
        //result = { make_float4(0, 0, 0, 0) };


        //half4Ushort4 result = { make_float4(t, t, t, t) };
        //surf2Dwrite<ushort4>(
        //    result.m_Ushort4,
        //    a_DepthOutPut,  //intput
        //    pixelX * sizeof(ushort4),
        //    pixelY,
        //    hipBoundaryModeTrap);

        float1 result = make_float1(t);
        surf2Dwrite<float1>(
            result,
            a_DepthOutPut,  //intput
            pixelX * sizeof(float1),
            pixelY,
            hipBoundaryModeTrap);

    }

}