#include "hip/hip_runtime.h"
#include "GPUShadingKernels.cuh"
#include <>

CPU_ON_GPU void MergeOutputChannels(
    const uint2 a_Resolution,
    const hipSurfaceObject_t a_Input,
    const hipSurfaceObject_t a_Output,
    const bool a_BlendOutput,
    const unsigned a_BlendCount
)
{
    const unsigned int pixelX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int pixelY = blockIdx.y * blockDim.y + threadIdx.y;

    constexpr unsigned int numChannels = static_cast<unsigned>(LightChannel::NUM_CHANNELS);

    if (pixelX < a_Resolution.x && pixelY < a_Resolution.y)
    {
        

        float4 mergedColor = { 0.f };
#pragma unroll 
        for(unsigned int channelIndex = 0; channelIndex < numChannels; ++channelIndex)
        {
			//Merge all channels except volumetric
			if (static_cast<LightChannel>(channelIndex) != LightChannel::VOLUMETRIC)
			{
				float4 channelColor{ 0.f };
				surf2DLayeredread<float4>(
					&channelColor,
					a_Input,
					pixelX * sizeof(float4),
					pixelY,
					channelIndex,
					hipBoundaryModeTrap);

				mergedColor += channelColor;
			}
        }
		//Blend volumetric
		float4 channelColor{ 0.f };
		surf2DLayeredread<float4>(
			&channelColor,
			a_Input,
			pixelX * sizeof(float4),
			pixelY,
			static_cast<unsigned>(LightChannel::VOLUMETRIC),
			hipBoundaryModeTrap);

		mergedColor = mergedColor * (1.0f - channelColor.w) + channelColor * channelColor.w;

        //If enabled, average between frames.
        if(a_BlendOutput)
        {
            float4 oldValue = { 0.f };

            surf2Dread<float4>(
                &oldValue,
                a_Output,
                pixelX * sizeof(float4),
                pixelY,
                hipBoundaryModeTrap);

            //Average results over the total blended frame count (so every frame counts just as much).
            float4 newValue = ((oldValue * static_cast<float>(a_BlendCount)) + mergedColor) / static_cast<float>(a_BlendCount + 1);
            surf2Dwrite<float4>(
                newValue,
                a_Output,
                pixelX * sizeof(float4),
                pixelY,
                hipBoundaryModeTrap);
            
        }
        //No blending so instead overwrite previous frame data.
        else
        {

            surf2Dwrite<float4>(
                mergedColor,
                a_Output,
                pixelX * sizeof(float4),
                pixelY,
                hipBoundaryModeTrap);

        }

    }
}