#include "hip/hip_runtime.h"
#include "SnapShotProcessing.cuh"

#include <>

CPU_ON_GPU void SeparateIntersectionRayBuffer(WaveFront::AtomicBuffer<WaveFront::IntersectionRayData>* a_IntersectionBuffer,
                                              float3* a_OriginBuffer, float3* a_DirectionBuffer, float3* a_ContributionBuffer)
{
    const uint32_t bufferSize = a_IntersectionBuffer->GetSize();
    const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t stride = blockDim.x * gridDim.x;

    for (uint32_t i = index; i < bufferSize - 1; i += stride)
    {
        WaveFront::IntersectionRayData* intersectionData = &a_IntersectionBuffer->data[index];

        a_OriginBuffer[intersectionData->m_PixelIndex] = intersectionData->m_Origin;
        a_DirectionBuffer[intersectionData->m_PixelIndex] = intersectionData->m_Direction;
        a_ContributionBuffer[intersectionData->m_PixelIndex] = intersectionData->m_Contribution;
    }
}

CPU_ON_GPU void SeparateMotionVectorBuffer(uint64_t a_BufferSize, WaveFront::MotionVectorBuffer* a_MotionVectorBuffer,
    float3* a_MotionVectorDirectionBuffer, float3* a_MotionVectorMagnitudeBuffer)
{
    const uint32_t bufferSize = a_BufferSize;
    const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t stride = blockDim.x * gridDim.x;

    for (uint32_t i = index; i < bufferSize - 1; i += stride)
    {
        const WaveFront::MotionVectorData motionVectorData = a_MotionVectorBuffer->m_MotionVectorBuffer[i];

        float2 velocity = motionVectorData.m_Velocity * 0.5f + 0.5f;
    	
        a_MotionVectorDirectionBuffer[i] = make_float3(velocity, 0.f);
        a_MotionVectorMagnitudeBuffer[i] = make_float3(length(motionVectorData.m_Velocity));
    }
}

CPU_ONLY void SeparateIntersectionRayBufferCPU(uint64_t a_BufferSize, WaveFront::AtomicBuffer<WaveFront::IntersectionRayData>* a_IntersectionBuffer,
    float3* a_OriginBuffer, float3* a_DirectionBuffer, float3* a_ContributionBuffer)
{
    const int blockSize = 256;
    const int numBlocks = (a_BufferSize + blockSize - 1) / blockSize;
    SeparateIntersectionRayBuffer<<<numBlocks, blockSize>>>(a_IntersectionBuffer, a_OriginBuffer, a_DirectionBuffer, a_ContributionBuffer);

	hipDeviceSynchronize();
};

CPU_ONLY void SeparateMotionVectorBufferCPU(uint64_t a_BufferSize, WaveFront::MotionVectorBuffer* a_MotionVectorBuffer,
    float3* a_MotionVectorDirectionBuffer, float3* a_MotionVectorMagnitudeBuffer)
{
    const int blockSize = 256;
    const int numBlocks = (a_BufferSize + blockSize - 1) / blockSize;
    SeparateMotionVectorBuffer<<<numBlocks, blockSize>>>(a_BufferSize, a_MotionVectorBuffer, a_MotionVectorDirectionBuffer, a_MotionVectorMagnitudeBuffer);

	hipDeviceSynchronize();
};